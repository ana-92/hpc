#include "hip/hip_runtime.h"

/* 
   Parámetros opcionales (en este orden): sumavectores  #size #blk
   #size: número de elementos en cada vector
   #blk: hilos por bloque CUDA
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
const int SIZE = 5376;    // Número predeterm. de elementos en los vectores
const int CUDA_BLK = 32;  // Número predeterm. hilos por bloque

/* 
   Para medir el tiempo transcurrido (elapsed time):

   resnfo: tipo de dato definido para abstraer la métrica de recursos a usar
   timenfo: tipo de dato definido para abstraer la métrica de tiempo a usar

   timestamp: abstrae función usada para tomar las muestras del tiempo transcurrido

   printtime: abstrae función usada para imprimir el tiempo transcurrido

   void myElapsedtime(resnfo start, resnfo end, timenfo *t): función para obtener 
   el tiempo transcurrido entre dos medidas
*/

#include <sys/time.h>
#include <sys/resource.h>

#ifdef _noWALL_
typedef struct rusage resnfo;
typedef struct _timenfo {
  double time;
  double systime;
} timenfo;
#define timestamp(sample) getrusage(RUSAGE_SELF, (sample))
#define printtime(t) printf("%15f s (%f user + %f sys) ",		\
			    t.time + t.systime, t.time, t.systime);
#else
typedef struct timeval resnfo;
typedef double timenfo;
#define timestamp(sample)     gettimeofday((sample), 0)
#define printtime(t) printf("%15f s ", t);
#endif

void myElapsedtime(const resnfo start, const resnfo end, timenfo *const t)
{
#ifdef _noWALL_
  t->time = (end.ru_utime.tv_sec + (end.ru_utime.tv_usec * 1E-6)) 
    - (start.ru_utime.tv_sec + (start.ru_utime.tv_usec * 1E-6));
  t->systime = (end.ru_stime.tv_sec + (end.ru_stime.tv_usec * 1E-6)) 
    - (start.ru_stime.tv_sec + (start.ru_stime.tv_usec * 1E-6));
#else
  *t = (end.tv_sec + (end.tv_usec * 1E-6)) 
    - (start.tv_sec + (start.tv_usec * 1E-6));
#endif /*_noWALL_*/
}



/**
 * Generar un vector con valores flotantes aleatorios
*/
void generar_vector( float *A, int N){
    srand(time(NULL));
    //Fue necesario linealizar el arreglo a A[nxn]
    for(int i=0; i< N; i++){
            for(int j=0;j<N; j++){
                A[N*i+j] = (rand( ) % 7501 ) / 1000.0f;
            }
        }
}

/**
 * Imprimir el valor de los vectores resultantes
*/
void imprimir_vectores (float *A,float *sum_seq, float *sum, int N){
    printf("\n");
    printf("****** Vector Result CPU ******\n");	
	for (int i = 0; i < N; ++i)
		printf("sum[%d] = %lf\n", i, sum_seq[i]);

    printf("\n");
    printf("****** Vector Result GPU ******\n");	
	for (int i = 0; i < N; ++i)
		printf("sum[%d] = %lf\n", i, sum[i]);
    
    printf("\n");

}

/**
 * Verificar si el vector generado en el CPU es igual al del GPU
*/
void comparar_vectores(float *sum_seq, float *sum, int N){
    int contador = 0;
    for (int i = 0; i < N; ++i){
        contador++;
        if(sum[i]!= sum_seq[i]){
            break;
        }
    }

    
    if(contador == N){
         printf("Los valores en cada arreglo son iguales\n");
    }
    else{
         printf("Los valores en cada arreglo NO son iguales\n");
    }   

}

/**
 * La suma del vector en el CPU
*/

void rowSums_seq(float* A, float* sum, int N){
    for(int i=0; i< N; i++){
        for(int j=0;j<N; j++){
                sum[i] += A[N*i+j];
            }
            
    }
}

/*
  Definición de nuestro kernel en CUDA
*/
__global__ void rowSums(float* A, float* sum, int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float tmpSum = 0;

    if (row < N){
        for (int k = 0; k< N; k++) {
            tmpSum += A[row * N + k] ;
        }
    }

    sum[row] = tmpSum;
              
 } 

int main(int argc, char *argv[])
{
    float *cA ,*cSum;
    
    // Para medir tiempos
    resnfo start, end;
    timenfo time;

    // Si existe mas de un argumento lo usamos sino utilizamos el default.
    int size = (argc > 1)?atoi (argv[1]):SIZE;
    int blk_size = (argc > 2)?atoi (argv[2]):CUDA_BLK;
    
    printf("Elementos en los arreglos =%d \n",size);
    printf("Hilos por bloque =%d \n",blk_size);

    int numBytesA = size*size*sizeof(float);
    int numBytesSum = size*sizeof(float);


    //Reserva de memoria en el CPU
    float *A = (float *) malloc(numBytesA);
    float *sum = (float *) malloc(numBytesSum);
    float *sum_seq = (float *) malloc(numBytesSum);

    //Generar vector de tamaño size*size (CPU)
    generar_vector(A,size);

    //Realizar la suma en el CPU
    timestamp(&start);
    rowSums_seq(A,sum_seq,size);
    timestamp(&end);

    myElapsedtime(start, end, &time);
    printtime(time);
    printf(" -> Sumar vectores en CPU \n");


    //Reserva de memoria en el GPU
    hipMalloc((void**)&cA, numBytesA);
    hipMalloc((void **) &cSum , numBytesSum) ;

    // CPU -> GPU
    hipMemcpy(cA, A, numBytesA, hipMemcpyHostToDevice);

    //Inicializar arreglo de sumas
    hipMemset(cSum, 0, numBytesSum);

    // Bloque unidimensional de hilos (*blk_size* hilos)
    dim3 dimBlock(blk_size);

    // Rejilla unidimensional (*ceil(n/blk_size)* bloques)
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);


	//Cálculo de la suma del vector en el GPU
    timestamp(&start);
    rowSums<<<dimGrid, dimBlock >>>(cA, cSum, size);
    timestamp(&end);

    myElapsedtime(start, end, &time);
    printtime(time);
    printf(" -> Sumar vectores en GPU \n");

    // GPU -> CPU
    hipMemcpy(sum, cSum, numBytesSum, hipMemcpyDeviceToHost); 

     //Imprimir vectores
    //imprimir_vectores(A,sum_seq,sum,size);

    //Comparar vectores
    comparar_vectores(sum_seq,sum,size);

	hipFree(cA);
	hipFree(cSum);

	return 0;
}